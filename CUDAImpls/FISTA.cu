
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <math.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__
void copy_array(int n, int m, float* x, float* y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    for (int i = row; i < n; i += (gridDim.x * blockDim.x))
    {
        for (int j = col; j < m; j += (gridDim.y * blockDim.y))
        {
            y[i*m+j] = x[i*m+j];
        }
    }
}

__global__
void copy_vector(int n, int *x, int *y)
{
    // x = y
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = y[i];
    }
}

__global__
void vector_minus(int n, float *x, float *y)
{
    // x = x - y
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = x[i] - y[i];
    }
}

__global__
void vector_multiply(int n, float k, float *x)
{
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = k * x[i];
    }
}

__global__
void shrink(int n, float *x, float l)
{
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        if (x[i] > 0)
        {
            x[i] = fmaxf(fabsf(x[i]) - l, 0);
        } else
        {
            x[i] = 0 - fmaxf(fabsf(x[i]) - l, 0);
        }
    }
}

int main(void)
{
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hiprandGenerator_t gen;
    hipsolverHandle_t cusolverH;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    int i, j;
    unsigned long long seed = 123456;
    // A is mxn matrix
    float *A, *AT;
    float *transpose;
    float *eigen;
    // x is R^n, b is R^m
    float *x, *b;
    int m = 32;
    int n = 16;
    int *devInfo;
    float *d_work;
    int lwork = 0;
    int info_gpu = 0;
    hipMallocManaged(&A, m*n*sizeof(float));
    hipMallocManaged(&AT, m*n*sizeof(float));
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&b, m*sizeof(float));
    hipMallocManaged(&transpose, n*n*sizeof(float));
    hipMallocManaged(&eigen, n*sizeof(float));
    hipMallocManaged(&devInfo, sizeof(int));
    hipDeviceSynchronize();
    hipMemset(&b, 0, m*sizeof(float));
    hipMemset(&transpose, 0, n*n*sizeof(float));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hipDeviceSynchronize();
    hiprandGenerateUniform(gen, A, m*n*sizeof(float));
    hiprandGenerateUniform(gen, x, n*sizeof(float));
    hipDeviceSynchronize();
    // cudaMemcpy(&AT, &A, m*n*sizeof(float), cudaMemcpyDefault);
    copy_array<<<dim3((m+15) / 16, (n+15) / 16), dim3(16, 16)>>>(m, n, A, AT);
    hipDeviceSynchronize();
    for (i = 0; i < n; i++)
    {
        printf("%f\n", x[i]);
    }
    printf("-------------------------\n");
    stat = hipblasCreate(&handle);
    float a = 1.0f;
    float c = 0.0f;
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &a, A, m, x, 1, &c, b, 1);
    hipDeviceSynchronize();
    // cuBLAS's explanation kind of misleading here. The leading dimension is before any operation
    stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &a, AT, m, A, m, &c, transpose, n);
    hipDeviceSynchronize();
    cusolver_status = hipsolverDnCreate(&cusolverH);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, n, transpose, n, eigen, &lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);
    cusolver_status = hipsolverDnSsyevd(cusolverH, jobz, uplo, n, transpose, n, eigen, d_work, lwork, devInfo);
    hipDeviceSynchronize();
    // for (i = 0; i < n; i++)
    // {
    //     printf("%f ", eigen[i]);
    // }
    // printf("\n");
    int iters = 1000;
    float step = 1 / (2 * eigen[n-1]);
    float l = 0.00001;
    float *x_est, *result, *temp;
    hipMallocManaged(&x_est, n*sizeof(float));
    hipMallocManaged(&result, m*sizeof(float));
    hipMallocManaged(&temp, n*sizeof(float));
    hipDeviceSynchronize();
    hipMemset(&x_est, 0, n*sizeof(float));
    for (int k = 0; k < iters; k++)
    {
        stat = hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &a, A, m, x_est, 1, &c, result, 1);
        hipDeviceSynchronize();
        vector_minus<<<1, m>>>(m, result, b);
        hipDeviceSynchronize();
        stat = hipblasSgemv(handle, HIPBLAS_OP_T, m, n, &a, A, m, result, 1, &c, temp, 1);
        hipDeviceSynchronize();
        vector_multiply<<<1, n>>>(n, 2*step, temp);
        hipDeviceSynchronize();
        vector_minus<<<1, n>>>(n, x_est, temp);
        hipDeviceSynchronize();
        shrink<<<1, n>>>(n, x_est, l * step);
        hipDeviceSynchronize();
    }
    for (j = 0; j < n; j++)
    {
        printf("%f\n", x_est[j]);
    }
    hipFree(A);
    hipFree(AT);
    hipFree(transpose);
    hipFree(eigen);
    hipFree(devInfo);
    hipFree(d_work);
    hipFree(x);
    hipFree(b);
    hipFree(x_est);
    hipFree(result);
    hipFree(temp);
    hipblasDestroy(handle);
    hiprandDestroyGenerator(gen);
}