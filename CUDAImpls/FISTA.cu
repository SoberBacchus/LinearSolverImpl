
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <math.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__
void copy_array(int n, int m, float* x, float* y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    for (int i = row; i < n; i += (gridDim.x * blockDim.x))
    {
        for (int j = col; j < m; j += (gridDim.y * blockDim.y))
        {
            y[i*m+j] = x[i*m+j];
        }
    }
}

__global__
void copy_vector(int n, int *x, int *y)
{
    // x = y
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = y[i];
    }
}

__global__
void vector_minus(int n, float *x, float *y)
{
    // x = x - y
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = x[i] - y[i];
    }
}

__global__
void vector_multiply(int n, float k, float *x)
{
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        x[i] = k * x[i];
    }
}

__global__
void shrink(int n, float *x, float l)
{
    for (int i = threadIdx.x; i < n; i += blockDim.x)
    {
        if (x[i] > 0)
        {
            x[i] = fmaxf(fabsf(x[i]) - l, 0);
        } else
        {
            x[i] = 0 - fmaxf(fabsf(x[i]) - l, 0);
        }
    }
}

float step_size(hipblasHandle_t handle, hipsolverHandle_t cusolverH, int m, int n, float *A)
{
    // A should be an mxn matrix;
    float *d_work;
    int lwork = 0;
    int *devInfo;
    float *eigen;
    float *AT;
    float *transpose;
    hipMallocManaged(&AT, m*n*sizeof(float));
    hipMallocManaged(&transpose, n*n*sizeof(float));
    hipMallocManaged(&eigen, n*sizeof(float));
    hipMallocManaged(&devInfo, sizeof(int));
    hipDeviceSynchronize();
    hipMemset(&transpose, 0, n*n*sizeof(float));
    hipDeviceSynchronize();
    copy_array<<<dim3((m+15) / 16, (n+15) / 16), dim3(16, 16)>>>(m, n, A, AT);
    hipDeviceSynchronize();
    float a = 1.0f;
    float c = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &a, AT, m, A, m, &c, transpose, n);
    hipDeviceSynchronize();
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverDnSsyevd_bufferSize(cusolverH, jobz, uplo, n, transpose, n, eigen, &lwork);
    hipMalloc((void**)&d_work, sizeof(float)*lwork);
    hipsolverDnSsyevd(cusolverH, jobz, uplo, n, transpose, n, eigen, d_work, lwork, devInfo);
    hipDeviceSynchronize();
    float step = 1 / (2 * eigen[n-1]);
    hipFree(eigen);
    hipFree(devInfo);
    hipFree(d_work);
    return step;
}

void ista(hipblasHandle_t handle, hipsolverHandle_t cusolverH, int m, int n, float *A, float *x, float *b, float *x_est, int iters, float l)
{
    float *result, *temp;
    hipMallocManaged(&result, m*sizeof(float));
    hipMallocManaged(&temp, n*sizeof(float));
    hipDeviceSynchronize();
    float step = step_size(handle, cusolverH, m, n, A);
    float a = 1.0f;
    float c = 0.0f;
    for (int k = 0; k < iters; k++)
    {
        hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &a, A, m, x_est, 1, &c, result, 1);
        hipDeviceSynchronize();
        vector_minus<<<1, m>>>(m, result, b);
        hipDeviceSynchronize();
        hipblasSgemv(handle, HIPBLAS_OP_T, m, n, &a, A, m, result, 1, &c, temp, 1);
        hipDeviceSynchronize();
        vector_multiply<<<1, n>>>(n, 2*step, temp);
        hipDeviceSynchronize();
        vector_minus<<<1, n>>>(n, x_est, temp);
        hipDeviceSynchronize();
        shrink<<<1, n>>>(n, x_est, l * step);
        hipDeviceSynchronize();
    }
    hipFree(result);
    hipFree(temp);
}

int main(void)
{
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    hiprandGenerator_t gen;
    hipsolverHandle_t cusolverH;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    int i, j;
    unsigned long long seed = 123456;
    // A is mxn matrix
    float *A;
    // x is R^n, b is R^m
    float *x, *b;
    int m = 32;
    int n = 16;
    int info_gpu = 0;
    hipMallocManaged(&A, m*n*sizeof(float));
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&b, m*sizeof(float));
    hipDeviceSynchronize();
    hipMemset(&b, 0, m*sizeof(float));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hipDeviceSynchronize();
    hiprandGenerateUniform(gen, A, m*n*sizeof(float));
    hiprandGenerateUniform(gen, x, n*sizeof(float));
    hipDeviceSynchronize();
    // cudaMemcpy(&AT, &A, m*n*sizeof(float), cudaMemcpyDefault);
    for (i = 0; i < n; i++)
    {
        printf("%f\n", x[i]);
    }
    printf("-------------------------\n");
    float a = 1.0f;
    float c = 0.0f;
    stat = hipblasCreate(&handle);
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &a, A, m, x, 1, &c, b, 1);
    hipDeviceSynchronize();
    cusolver_status = hipsolverDnCreate(&cusolverH);
    int iters = 8000;
    float l = 0.00001;
    float *x_est;
    hipMallocManaged(&x_est, n*sizeof(float));
    hipDeviceSynchronize();
    hipMemset(&x_est, 0, n*sizeof(float));
    ista(handle, cusolverH, m, n, A, x, b, x_est, iters, l);
    for (j = 0; j < n; j++)
    {
        printf("%f\n", x_est[j]);
    }
    hipFree(A);
    hipFree(x);
    hipFree(b);
    hipFree(x_est);
    hipblasDestroy(handle);
    hiprandDestroyGenerator(gen);
}